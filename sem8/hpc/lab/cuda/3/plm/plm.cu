#include "./common/helpers.h"

#define SIZE (10 * 1024 * 1024)

float cuda_malloc_test(int size, bool up) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    a = (int*)malloc(size * sizeof(*a)); 

    HANDLE_NULL(a);
    HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

    HANDLE_ERROR(hipEventRecord(start, 0)); 

    for (int i = 0; i < 100; i++) {
        if (up) {
            HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
        } else {
            HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
        }
    }

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    free(a);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime; 
}

int main(void) {
    float elapsedTime;
    float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

    elapsedTime = cuda_malloc_test(SIZE, true);

    printf("Total time for copy up: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy up:  %3.1f\n", MB / (elapsedTime / 1000));

    elapsedTime = cuda_malloc_test(SIZE, false); 
    
    printf("Total time for copy down: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy down:  %3.1f\n", MB / (elapsedTime / 1000));

    return 0;
}
